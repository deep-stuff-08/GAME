#include "hip/hip_runtime.h"
/*
    GPU kernel functions runs on the device
*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "..\..\include\huffman.cuh"
/*
* Compression function
* Single run and no overflow
*/
__global__ void compress(unsigned char * device_inputFileData,
                         unsigned int * device_compressedDataOffset,
                         struct huffmanDictionary * device_huffmanDictionary,
                         unsigned char * device_byteCompressedData,
                         unsigned int device_inputFileLength,
                         unsigned int constMemoryFlag)
{
    __shared__ struct huffmanDictionary table;
    memcpy(& table, device_huffmanDictionary, sizeof(struct huffmanDictionary));
    unsigned int inputFileLength = device_inputFileLength;
    unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;

    // when shared memory is sufficient
    if(constMemoryFlag == 0){
        for(int i = pos; i < inputFileLength; i += blockDim.x){
            for(int k = 0; k < table.bitSequenceLength[device_inputFileData[i]]; k++){
                device_byteCompressedData[device_compressedDataOffset[i] + k] = table.bitSequence[device_inputFileData[i]][k];
            }
        }
    }

    // use the shared and constant memory
    else {
        for(int i = pos; i < inputFileLength; i += blockDim.x){
            for(int k = 0; k < table.bitSequenceLength[device_inputFileData[i]]; k++){

                if(k < 191){
                    device_byteCompressedData[device_compressedDataOffset[i] + k] = table.bitSequence[device_inputFileData[i]][k];
                } else {
                    device_byteCompressedData[device_compressedDataOffset[i] + k] = device_bitSequenceConstMemory[device_inputFileData[i]][k];
                }
            }
        }
    }

    __syncthreads();

    for(int i = pos * 8; i < device_compressedDataOffset[inputFileLength]; i += blockDim.x * 8){
        for(int j = 0; j < 8; j++){
            if(device_byteCompressedData[i + j] == 0){
                device_inputFileData[i / 8] = device_inputFileData[i / 8] << 1;
            } else {
                device_inputFileData[i / 8] = (device_inputFileData[i / 8] << 1) | 1;
            }
        }
    }
}

/*
* Compression function
* Single run with overflow
*/
__global__ void compress(unsigned char * device_inputFileData,
                         unsigned int * device_compressedDataOffset,
                         struct huffmanDictionary * device_huffmanDictionary,
                         unsigned char * device_byteCompressedData,
                         unsigned char * device_tempOverflow,
                         unsigned int device_inputFileLength,
                         unsigned int constMemoryFlag,
                         unsigned int overflowPosition)
{
    __shared__ struct huffmanDictionary table;
    memcpy(& table, device_huffmanDictionary, sizeof(struct huffmanDictionary));
    unsigned int inputFileLength = device_inputFileLength;
    unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int offsetOverflow;

    // when shared memory is sufficient
    if(constMemoryFlag == 0){
        for(int i = pos; i < overflowPosition; i += blockDim.x){
            for(int k = 0; k < table.bitSequenceLength[device_inputFileData[i]]; k++){
                device_byteCompressedData[device_compressedDataOffset[i] + k] = table.bitSequence[device_inputFileData[i]][k];
            }
        }
        for(int i = overflowPosition + pos; i < inputFileLength - 1; i += blockDim.x){
            for(int k = 0; k < table.bitSequenceLength[device_inputFileData[i + 1]]; k++){
                device_tempOverflow[device_compressedDataOffset[i + 1] + k] = table.bitSequence[device_inputFileData[i + 1]][k];
            }
        }
        if(pos == 0){
            memcpy(& device_tempOverflow[device_compressedDataOffset[(overflowPosition + 1)] - table.bitSequenceLength[device_inputFileData[overflowPosition]]],
                   & table.bitSequenceLength[device_inputFileData[overflowPosition]], table.bitSequenceLength[device_inputFileData[overflowPosition]]);
        }
    }

    // use the shared and constant memory
    else {
        for(int i = pos; i < overflowPosition; i += blockDim.x){
            for(int k = 0; k < table.bitSequenceLength[device_inputFileData[i]]; k++){

                if(k < 191){
                    device_byteCompressedData[device_compressedDataOffset[i] + k] = table.bitSequence[device_inputFileData[i]][k];
                } else {
                    device_byteCompressedData[device_compressedDataOffset[i] + k] = device_bitSequenceConstMemory[device_inputFileData[i]][k];
                }
            }
        }
        for(int i = overflowPosition + pos; i < inputFileLength - 1; i += blockDim.x){
            for(int k = 0; k < table.bitSequenceLength[device_inputFileData[i + 1]]; k++){

                if(k < 191){
                    device_tempOverflow[device_compressedDataOffset[i + 1] + k] = table.bitSequence[device_inputFileData[i + 1]][k];
                } else {
                    device_tempOverflow[device_compressedDataOffset[i + 1] + k] = device_bitSequenceConstMemory[device_inputFileData[i + 1]][k];
                }
            }
        }
        if(pos == 0){
            memcpy(& device_tempOverflow[device_compressedDataOffset[(overflowPosition + 1)] - table.bitSequenceLength[device_inputFileData[overflowPosition]]],
                   & table.bitSequenceLength[device_inputFileData[overflowPosition]], table.bitSequenceLength[device_inputFileData[overflowPosition]]);
        }
    }

    __syncthreads();

    for(int i = pos * 8; i < device_compressedDataOffset[inputFileLength]; i += blockDim.x * 8){
        for(int j = 0; j < 8; j++){
            if(device_byteCompressedData[i + j] == 0){
                device_inputFileData[i / 8] = device_inputFileData[i / 8] << 1;
            } else {
                device_inputFileData[i / 8] = (device_inputFileData[i / 8] << 1) | 1;
            }
        }
    }

    offsetOverflow = device_compressedDataOffset[overflowPosition] / 8;

    for(int i = pos * 8; i < device_compressedDataOffset[inputFileLength]; i += blockDim.x * 8){
        for(int j = 0; j < 8; j++){
            if(device_tempOverflow[i + j] == 0){
                device_inputFileData[(i / 8) + offsetOverflow] = device_inputFileData[(i / 8) + offsetOverflow] << 1;
            } else {
                device_inputFileData[(i / 8) + offsetOverflow] = (device_inputFileData[(i / 8) + offsetOverflow] << 1) | 1;
            }
        }
    }
}

/*
* Compression function
* Multiple run with no overflow
*/
__global__ void compress(unsigned char * device_inputFileData,
                         unsigned int * device_compressedDataOffset,
                         struct huffmanDictionary * device_huffmanDictionary,
                         unsigned char * device_byteCompressedData,
                         unsigned int device_lowerPosition,
                         unsigned int constMemoryFlag,
                         unsigned int device_upperPosition)
{
    __shared__ struct huffmanDictionary table;
    memcpy(& table, device_huffmanDictionary, sizeof(struct huffmanDictionary));
    unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;

    // when shared memory is sufficient
    if(constMemoryFlag == 0){
        for(int i = pos + device_lowerPosition; i < device_upperPosition; i += blockDim.x){
            for(int k = 0; k < table.bitSequenceLength[device_inputFileData[i]]; k++){
                device_byteCompressedData[device_compressedDataOffset[i] + k] = table.bitSequence[device_inputFileData[i]][k];
            }
        }
        if(pos == 0 && device_lowerPosition != 0){
            memcpy(& device_byteCompressedData[device_compressedDataOffset[device_lowerPosition] - table.bitSequenceLength[device_inputFileData[device_lowerPosition - 1]]],
                   & table.bitSequenceLength[device_inputFileData[device_lowerPosition - 1]], table.bitSequenceLength[device_inputFileData[device_lowerPosition - 1]]);
        }
    }

    // use shared and constant memory
    else {
        for(int i = pos + device_lowerPosition; i < device_upperPosition; i += blockDim.x){
            for(int k = 0; k < table.bitSequenceLength[device_inputFileData[i]]; k++){

                if(k < 191){
                    device_byteCompressedData[device_compressedDataOffset[i] + k] = table.bitSequence[device_inputFileData[i]][k];
                } else {
                    device_byteCompressedData[device_compressedDataOffset[i] + k] = device_bitSequenceConstMemory[device_inputFileData[i]][k];
                }
            }
        }
        if(pos == 0 && device_lowerPosition != 0){
            memcpy(& device_byteCompressedData[device_compressedDataOffset[device_lowerPosition] - table.bitSequenceLength[device_inputFileData[device_lowerPosition - 1]]],
                   & table.bitSequenceLength[device_inputFileData[device_lowerPosition - 1]], table.bitSequenceLength[device_inputFileData[device_lowerPosition - 1]]);
        }
    }

    __syncthreads();

    for(int i = pos * 8; i < device_compressedDataOffset[device_upperPosition]; i += blockDim.x * 8){
        for(int j = 0; j < 8; j++){
            if(device_byteCompressedData[i + j] == 0){
                device_inputFileData[(i / 8)] = device_inputFileData[(i / 8)] << 1;
            } else {
                device_inputFileData[(i / 8)] = (device_inputFileData[(i / 8)] << 1) | 1;
            }
        }
    }
}

/*
* Compression function
* Multiple run with overflow
*/
__global__ void compress(unsigned char * device_inputFileData,
                         unsigned int * device_compressedDataOffset,
                         struct huffmanDictionary * device_huffmanDictionary,
                         unsigned char * device_byteCompressedData,
                         unsigned char * device_tempOverflow,
                         unsigned int device_lowerPosition,
                         unsigned int constMemoryFlag,
                         unsigned int device_upperPosition,
                         unsigned int overflowPosition)
{
    __shared__ struct huffmanDictionary table;
    memcpy(& table, device_huffmanDictionary, sizeof(struct huffmanDictionary));
    unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int offsetOverflow;

    // when shared memory is sufficient
    if(constMemoryFlag == 0){
        for(int i = pos + device_lowerPosition; i < overflowPosition; i += blockDim.x){
            for(int k = 0; k < table.bitSequenceLength[device_inputFileData[i]]; k++){
                device_byteCompressedData[device_compressedDataOffset[i] + k] = table.bitSequence[device_inputFileData[i]][k];
            }
        }
        for(int i = pos + overflowPosition; i < device_upperPosition - 1; i += blockDim.x){
            for(int k = 0; k < table.bitSequenceLength[device_inputFileData[i + 1]]; k++){
                device_tempOverflow[device_compressedDataOffset[i + 1] + k] = table.bitSequence[device_inputFileData[i + 1]][k];
            }
        }

        if(pos == 0){
            memcpy(& device_tempOverflow[device_compressedDataOffset[overflowPosition + 1] - table.bitSequenceLength[device_inputFileData[overflowPosition]]],
                   & table.bitSequenceLength[device_inputFileData[overflowPosition - 1]], table.bitSequenceLength[device_inputFileData[overflowPosition]]);
        }
        if(pos == 0 && device_lowerPosition != 0){
            memcpy(& device_byteCompressedData[device_compressedDataOffset[device_lowerPosition] - table.bitSequenceLength[device_inputFileData[device_lowerPosition - 1]]],
                   & table.bitSequenceLength[device_inputFileData[device_lowerPosition - 1]], table.bitSequenceLength[device_inputFileData[device_lowerPosition - 1]]);
        }
    }

    // use shared and constant memory
    else {
        for(int i = pos + device_lowerPosition; i < device_upperPosition; i += blockDim.x){
            for(int k = 0; k < table.bitSequenceLength[device_inputFileData[i]]; k++){

                if(k < 191){
                    device_byteCompressedData[device_compressedDataOffset[i] + k] = table.bitSequence[device_inputFileData[i]][k];
                } else {
                    device_byteCompressedData[device_compressedDataOffset[i] + k] = device_bitSequenceConstMemory[device_inputFileData[i]][k];
                }
            }
        }
        for(int i = overflowPosition + pos; i < device_upperPosition - 1; i += blockDim.x){
            for(int k = 0; k < table.bitSequenceLength[device_inputFileData[i + 1]]; k++){

                if(k < 191){
                    device_tempOverflow[device_compressedDataOffset[i] + k] = table.bitSequence[device_inputFileData[i]][k];
                } else {
                    device_tempOverflow[device_compressedDataOffset[i] + k] = device_bitSequenceConstMemory[device_inputFileData[i]][k];
                }
            }
        }
        if(pos == 0){
            memcpy(& device_tempOverflow[device_compressedDataOffset[(overflowPosition + 1)] - table.bitSequenceLength[device_inputFileData[overflowPosition]]],
                   & table.bitSequenceLength[device_inputFileData[overflowPosition]], table.bitSequenceLength[device_inputFileData[overflowPosition]]);
        }
        if(pos == 0 && device_lowerPosition != 0){
            memcpy(& device_byteCompressedData[device_compressedDataOffset[device_lowerPosition] - table.bitSequenceLength[device_inputFileData[device_lowerPosition - 1]]],
                   & table.bitSequenceLength[device_inputFileData[device_lowerPosition - 1]], table.bitSequenceLength[device_inputFileData[device_lowerPosition - 1]]);
        }
    }

    __syncthreads();

    for(int i = pos * 8; i < device_compressedDataOffset[overflowPosition]; i += blockDim.x * 8){
        for(int j = 0; j < 8; j++){
            if(device_byteCompressedData[i + j] == 0){
                device_inputFileData[i / 8] = device_inputFileData[i / 8] << 1;
            } else {
                device_inputFileData[i / 8] = (device_inputFileData[i / 8] << 1) | 1;
            }
        }
    }

    offsetOverflow = device_compressedDataOffset[overflowPosition] / 8;

    for(int i = pos * 8; i < device_compressedDataOffset[device_upperPosition]; i += blockDim.x * 8){
        for(int j = 0; j < 8; j++){
            if(device_tempOverflow[i + j] == 0){
                device_inputFileData[(i / 8) + offsetOverflow] = device_inputFileData[(i / 8) + offsetOverflow] << 1;
            } else {
                device_inputFileData[(i / 8) + offsetOverflow] = (device_inputFileData[(i / 8) + offsetOverflow] << 1) | 1;
            }
        }
    }
}