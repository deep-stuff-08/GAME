#include"cuhelper.cuh"
#include<stdio.h>

float* createCUDABuffer(size_t s, void* data) {
	hipError_t err;
	float *cu;
	err = hipMalloc(&cu, s);
	if(err != hipSuccess) {
		printf("%d at %d\n", err, __LINE__);
	}

	if(data != NULL) {
		err = hipMemcpy(cu, data, s, hipMemcpyHostToDevice);
		if(err != hipSuccess) {
			printf("%d at %d\n", err, __LINE__);
		}
	}
	return cu;
}

void getCudaData(float* buffer, size_t s, float** data) {
	hipError_t err = hipMemcpy(*data, buffer, s, hipMemcpyDeviceToHost);
	if(err != hipSuccess) {
		printf("%d at %d\n", err, __LINE__);
	}
}