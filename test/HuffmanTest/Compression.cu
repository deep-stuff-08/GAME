#include "hip/hip_runtime.h"
/*
* Compression using the parallel functions and GPU
*/
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <limits.h>
#include "..\..\src\include\huffman.cuh"

#define BLOCK_SIZE 1024
#define MIN_SCRATCH_SIZE 50 * 1024 * 1024
#define DEBUG 1

struct huffmanNode* huffmanTreeNode_head;
struct huffmanDictionary huffmanDictionary;
struct huffmanNode huffmanTreeNode[512];
unsigned char bitSequenceConstMemory[256][255];
unsigned int constMemoryFlag = 0;

int main(int argc, char** argv) {

    unsigned int distinctCharacterCount, mergedHuffmanNodes, inputFileLength;
    unsigned int frequency[256];
    unsigned char* inputFileData, bitSequenceLength = 0, bitSequence[255];
    unsigned int* compressedDataOffset, cpuTimeUsed;
    unsigned int integerOverflowFlag;
    
    size_t memFree, memTotal, memRequired, memOffset, memData;
    int numberOfKernels;
    clock_t start, end;

    FILE* inputFile, * compressedFile;

    // check the arguments
    if (argc != 3) {
        printf("Arguments should be input file and output file");
        return -1;
    }
    // read input file, get length and data
    inputFile = fopen(argv[1], "rb");
    fseek(inputFile, 0, SEEK_END);
    inputFileLength = ftell(inputFile);
    fseek(inputFile, 0, SEEK_SET);
    inputFileData = (unsigned char*)malloc(inputFileLength * sizeof(unsigned char));
    fread(inputFileData, sizeof(unsigned char), inputFileLength, inputFile);
    fclose(inputFile);

    // starting the clock, tick tick
    start = clock();

    // find frequency of each symbols
    for (int i = 0; i < 256; i++)
        frequency[i] = 0;
    for (int i = 0; i < inputFileLength; i++)
        frequency[inputFileData[i]] ++;

    // initialize the nodes
    distinctCharacterCount = 0;
    for (int i = 0; i < 256; i++) {
        if (frequency[i] > 0) {
            huffmanTreeNode[distinctCharacterCount].frequency = frequency[i];
            huffmanTreeNode[distinctCharacterCount].letter = i;
            huffmanTreeNode[distinctCharacterCount].left = NULL;
            huffmanTreeNode[distinctCharacterCount].right = NULL;
            distinctCharacterCount++;
        }
    }

    // build the huffman tree
    for (int i = 0; i < distinctCharacterCount - 1; i++) {
        mergedHuffmanNodes = 2 * i;
        sortHuffmanTree(i, distinctCharacterCount, mergedHuffmanNodes);
        buildHuffmanTree(i, distinctCharacterCount, mergedHuffmanNodes);
    }
    if (distinctCharacterCount == 1) {
        huffmanTreeNode_head = &huffmanTreeNode[0];
    }

    // build the huffman dictionary
    buildHuffmanDictionary(huffmanTreeNode_head, bitSequence, bitSequenceLength);

    // calculating memory requirements
    // gpu memory
    hipMemGetInfo(&memFree, &memTotal);

    // debug
    if (DEBUG)
        printf("Free Memory :: %zu \n", memFree);

    // offset array requirements
    memOffset = 0;
    for (int i = 0; i < 256; i++)
        memOffset += frequency[i] * huffmanDictionary.bitSequenceLength[i];
    memOffset = memOffset % 8 == 0 ? memOffset : memOffset + 8 - memOffset % 8;

    // other memory requirements
    memData = inputFileLength + (inputFileLength + 1) * sizeof(unsigned int) + sizeof(huffmanDictionary);

    if (memFree - memData < MIN_SCRATCH_SIZE) {
        printf("\nExiting not enough memory on GPU :: \nMem Free :: %zu\nMin Required :: %zu\n", memFree, memData + MIN_SCRATCH_SIZE);
        return -1;
    }

    memRequired = memFree - memData - 10 * 1024 * 1024;
    numberOfKernels = ceil((double)memOffset / memRequired);
    integerOverflowFlag = memRequired + 255 <= UINT_MAX || memOffset + 255 <= UINT_MAX ? 0 : 1;

    if (DEBUG) {
        printf("\n\tInput File Size :: %u\n\tOutput Size :: %zu\n\tNumber of Kernels :: %d\n\tInteger Overflow flag :: %d\n", inputFileLength, memOffset / 8, numberOfKernels, integerOverflowFlag);
    }

    // generate offset data array
    compressedDataOffset = (unsigned int*)malloc((inputFileLength + 1) * sizeof(unsigned int));

    // launch kernel
    launchCudaHuffmanCompress(inputFileData, compressedDataOffset, inputFileLength, numberOfKernels, integerOverflowFlag, memRequired);

    // end the clock, tick tick
    end = clock();

    // writing the compressed file to the output
    compressedFile = fopen(argv[2], "wb");
    fwrite(&inputFileLength, sizeof(unsigned int), 1, compressedFile);
    fwrite(frequency, sizeof(unsigned int), 256, compressedFile);
    fwrite(inputFileData, sizeof(unsigned char), memOffset / 8, compressedFile);
    fclose(compressedFile);

    cpuTimeUsed = ((end - start)) * 1000 / CLOCKS_PER_SEC;
    printf("\n\nTime taken :: %d:%d s\n", cpuTimeUsed / 1000, cpuTimeUsed % 1000);

    free(inputFileData);
    free(compressedDataOffset);

    return 0;
}